#include "utils.h"

__device__ vec3 RandomInUnitSphere(hiprandState* localRandState)
{
	vec3 p;
	do 
	{
		p = 2.0f * vec3(hiprand_uniform(localRandState), hiprand_uniform(localRandState), hiprand_uniform(localRandState)) 
			- vec3(1.0f, 1.0f, 1.0f); // in the range of [-1, 1]
	} while (p.LengthSquared() >= 1.0f);

	return p;
}

void CheckCuda(hipError_t result, const char* func, const char* filepath, const uint32_t line)
{
	if (result)
	{
		std::cerr << "CUDA::ERROR:" << static_cast<uint32_t>(result) << " in file: \"" << filepath
			<< "\": line " << line << " - '" << func << "'" << std::endl;
		hipDeviceReset();
		__debugbreak();
	}
}

int32_t InitCudaDevice()
{
	int32_t dev;
	hipDeviceProp_t prop;

	memset(&prop, 0, sizeof(hipDeviceProp_t));
	// Compute capability >= 3.0
	prop.major = 3;
	prop.minor = 0;
	cudaCheckErrors(hipChooseDevice(&dev, &prop)); // Choose a device with compute capability >= 3.0
	cudaCheckErrors(cudaGLSetGLDevice(dev));

	return dev;
}

void InitCudaTexture(hipGraphicsResource_t& textureResource, hipResourceDesc& resourceDesc, uint32_t textureID)
{
	// Register texture with CUDA resource
	cudaCheckErrors(hipGraphicsGLRegisterImage(&textureResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
	memset(&resourceDesc, 0, sizeof(resourceDesc));
	resourceDesc.resType = hipResourceTypeArray;
}
