#include "utils.h"

void CheckCuda(hipError_t result, const char* func, const char* filepath, const uint32_t line)
{
	if (result)
	{
		std::cerr << "CUDA::ERROR:" << static_cast<uint32_t>(result) << " in file: \"" << filepath
			<< "\": line " << line << " - '" << func << "'" << std::endl;
		hipDeviceReset();
		__debugbreak();
	}
}

int32_t InitCudaDevice()
{
	int32_t dev;
	hipDeviceProp_t prop;

	memset(&prop, 0, sizeof(hipDeviceProp_t));
	// Compute capability >= 3.0
	prop.major = 3;
	prop.minor = 0;
	cudaCheckErrors(hipChooseDevice(&dev, &prop)); // Choose a device with compute capability >= 3.0
	cudaCheckErrors(cudaGLSetGLDevice(dev));

	return dev;
}

void InitCudaTexture(hipGraphicsResource_t& textureResource, hipResourceDesc& resourceDesc, uint32_t textureID)
{
	// Register texture with CUDA resource
	cudaCheckErrors(hipGraphicsGLRegisterImage(&textureResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
	memset(&resourceDesc, 0, sizeof(resourceDesc));
	resourceDesc.resType = hipResourceTypeArray;
}
