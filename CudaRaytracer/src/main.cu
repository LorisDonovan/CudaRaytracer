#include "hip/hip_runtime.h"
#include <iostream>

#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

#include "opengl/windowInit.h"
#include "opengl/screen.h"

#include "render/ray.h"
#include "render/hittable.h"
#include "render/hittableList.h"
#include "render/sphere.h"
#include "render/camera.h"

#include "utils/vec3.h"
#include "utils/timer.h"
#include "utils/utils.h"


// Settings
constexpr int32_t numSamples = 32;
constexpr float aspectRatio = 16.0f / 9.0f;
const  uint32_t height = 540;
const  uint32_t width  = static_cast<uint32_t>(height * aspectRatio);


// ----------Raytracer-------------------------------------
__device__ vec3 RayColor(const Ray& ray, Hittable** hittable);
__global__ void RenderInit(hiprandState* randState);
__global__ void Render(hipSurfaceObject_t surfaceObj, Hittable** world, Camera** cam, hiprandState* randState);
__global__ void CreateWorld(Camera** cam, Hittable** list, Hittable** world);
__global__ void FreeWorld(Camera** cam, Hittable** list, Hittable** world);


int main(int argc, char** argv)
{
	// CUDA resources
	hipResourceDesc       resourceDesc;
	hipGraphicsResource_t textureResource;
	hipArray_t            textureArray;
	hipSurfaceObject_t    surfaceObj = 0;

	// Initialize opengl and cuda interop
	GLFWwindow* window = InitWindow(width, height);
	int32_t cudaDevID  = InitCudaDevice();

	// Initialize vertex array and vertex buffer
	uint32_t quadVA, quadVB, shaderID;
	InitFbQuad(quadVA, quadVB, shaderID);

	// Initialize texture
	uint32_t textureID = InitGLTexture(width, height);
	InitCudaTexture(textureResource, resourceDesc, textureID);
	
	// Create Scene objects
	Camera** d_Cam;
	cudaCheckErrors(hipMalloc((void**)&d_Cam, sizeof(Camera*)));
	Hittable** d_List;
	cudaCheckErrors(hipMalloc((void**)&d_List, 2 * sizeof(Hittable*)));
	Hittable** d_World;
	cudaCheckErrors(hipMalloc((void**)&d_World, sizeof(Hittable*)));
	CreateWorld<<<1, 1>>>(d_Cam, d_List, d_World);
	cudaCheckErrors(hipGetLastError());
	cudaCheckErrors(hipDeviceSynchronize());

	// CUDA kernel thread layout
	int32_t numThreads = 32;
	dim3 blocks((width + numThreads - 1) / numThreads, (height + numThreads - 1) / numThreads);
	dim3 threads(numThreads, numThreads);

	// Initialize random numbers for Rendering
	hiprandState* d_RandState;
	{
		Timer t;
		cudaCheckErrors(hipMalloc((void**)&d_RandState, width * height * sizeof(hiprandState)));
		RenderInit<<<blocks, threads>>>(d_RandState);
		cudaCheckErrors(hipGetLastError());
		cudaCheckErrors(hipDeviceSynchronize());
	}
	// Call Render function
	{
		Timer t; // Starts a timer when created and stops when destroyed
		// CUDA register and create surface object resource
		cudaCheckErrors(hipGraphicsMapResources(1, &textureResource));
		cudaCheckErrors(hipGraphicsSubResourceGetMappedArray(&textureArray, textureResource, 0, 0));
		resourceDesc.res.array.array = textureArray;
		cudaCheckErrors(hipCreateSurfaceObject(&surfaceObj, &resourceDesc));
		Render<<<blocks, threads>>>(surfaceObj, d_World, d_Cam, d_RandState);
		cudaCheckErrors(hipGraphicsUnmapResources(1, &textureResource)); // sync cuda operations before graphics calls
		cudaCheckErrors(hipGetLastError());
		cudaCheckErrors(hipDeviceSynchronize());
	}

	while (!glfwWindowShouldClose(window))
	{
		// Render
		glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);
		ShaderBind(shaderID);
		glBindVertexArray(quadVA);
		TextureBind(textureID);
		glDrawArrays(GL_TRIANGLES, 0, 6);

		glfwSwapBuffers(window);
		glfwPollEvents();
	}

	// Cleanup
	FreeWorld<<<1, 1>>>(d_Cam, d_List, d_World);
	cudaCheckErrors(hipGetLastError());

	cudaCheckErrors(hipFree(d_List));
	cudaCheckErrors(hipFree(d_World));
	cudaCheckErrors(hipFree(d_Cam));
	Cleanup(quadVA, quadVB, textureID, shaderID);
	glfwTerminate();
	return 0;
}


// ----------Raytracer-------------------------------------
__global__ void RenderInit(hiprandState* randState)
{
	int32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	int32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= width || y >= height)
		return;

	int32_t pixelIdx = x + y * width;
	// Random numbers for each thread
	hiprand_init(1984, pixelIdx, 0, &randState[pixelIdx]);
}

__global__ void Render(hipSurfaceObject_t surfaceObj, Hittable** world, Camera** cam, hiprandState* randState)
{
	int32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	int32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if ((x >= width) || (y >= height))
		return;

	int32_t pixelIdx = x + y * width;
	hiprandState localRandState = randState[pixelIdx];
	vec3 color(0.0f, 0.0f, 0.0f);
	for (int i = 0; i < numSamples; i++)
	{
		// Offset values to move the ray across the screen
		float u = float(x + hiprand_uniform(&localRandState)) / float(width);
		float v = float(y + hiprand_uniform(&localRandState)) / float(height);
		color  += RayColor((*cam)->GetRay(u, v), world);
	}
	
	// Calculate color
	color     /= float(numSamples);
	uint8_t r  = uint8_t(color.r() * 255);
	uint8_t g  = uint8_t(color.g() * 255);
	uint8_t b  = uint8_t(color.b() * 255);

	uchar4 data = make_uchar4(r, g, b, 255);
	surf2Dwrite(data, surfaceObj, x * sizeof(uchar4), y);
}

__device__ vec3 RayColor(const Ray& ray, Hittable** world)
{
	HitRecords rec;
	if ((*world)->Hit(ray, 0.001f, inf, rec))
		return 0.5f * vec3(rec.Normal + vec3(1.0f, 1.0f, 1.0f)); // Mapping to [0, 1]

	vec3 dir = ray.GetDirection();        // Direction of ray is a unit vector
	float t  = 0.5f * (dir.y() + 1.0f);   // Mapping y in the range [0, 1]
	return (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f); // Blend the background from blue to white vertically
}

__global__ void CreateWorld(Camera** cam, Hittable** list, Hittable** world)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		list[0] = new Sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f);
		list[1] = new Sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f);
		*world = new HittableList(list, 2);
		*cam = new Camera(aspectRatio);
	}
}

__global__ void FreeWorld(Camera** cam, Hittable** list, Hittable** world)
{
	delete list[0];
	delete list[1];
	delete* world;
	delete* cam;
}


